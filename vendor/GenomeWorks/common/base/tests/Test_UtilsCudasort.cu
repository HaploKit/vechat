#include "hip/hip_runtime.h"
/*
* Copyright 2019-2020 NVIDIA CORPORATION.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include "gtest/gtest.h"

#include <algorithm>
#include <numeric>
#include <random>
#include <vector>

#include <thrust/host_vector.h>

#include <claraparabricks/genomeworks/utils/cudasort.cuh>
#include <claraparabricks/genomeworks/utils/device_buffer.hpp>
#include <claraparabricks/genomeworks/utils/signed_integer_utils.hpp>

namespace claraparabricks
{

namespace genomeworks
{

template <typename MoreSignificantKeyT,
          typename LessSignificantKeyT,
          typename ValueT>
void test_function(device_buffer<MoreSignificantKeyT>& more_significant_keys,
                   device_buffer<LessSignificantKeyT>& less_significant_keys,
                   device_buffer<ValueT>& input_values,
                   const MoreSignificantKeyT max_value_of_more_significant_key,
                   const LessSignificantKeyT max_value_of_less_significant_key)
{
    ASSERT_EQ(get_size(input_values), get_size(more_significant_keys));
    ASSERT_EQ(get_size(input_values), get_size(less_significant_keys));

    cudautils::sort_by_two_keys(more_significant_keys,
                                less_significant_keys,
                                input_values,
                                max_value_of_more_significant_key,
                                max_value_of_less_significant_key);

    thrust::host_vector<ValueT> sorted_values_h(input_values.size());
    cudautils::device_copy_n(input_values.data(), input_values.size(), sorted_values_h.data()); // D2H

    ASSERT_EQ(get_size(sorted_values_h), get_size(input_values));
    // sort is done by two keys and not values, but tests cases are intentionally made so the values are sorted as well
    for (typename device_buffer<ValueT>::size_type i = 1; i < input_values.size(); ++i)
    {
        EXPECT_LE(sorted_values_h[i - 1], sorted_values_h[i]) << "index: " << i << std::endl;
    }
}

// repreat this test with differnt combinations of types, more significant key has larger max value than less significant key
template <typename MoreSignificantKeyT,
          typename LessSignificantKeyT,
          typename ValueT>
void short_test_template_larger_more_significant_key()
{
    // more less value
    //   60   1   610
    //   20   4   240
    //   50   5   550
    //   40   2   420
    //   40   5   450
    //   20   1   210
    //   20   2   220
    //   30   8   380
    //   30   7   370
    //   50   1   510
    //   50   3   530
    //   40   5   451
    //   80   4   840
    const std::vector<MoreSignificantKeyT> more_significant_keys_vec = {60, 20, 50, 40, 40, 20, 20, 30, 30, 50, 50, 40, 80};
    const std::vector<LessSignificantKeyT> less_significant_keys_vec = {1, 4, 5, 2, 5, 1, 2, 8, 7, 1, 3, 5, 4};
    const std::vector<ValueT> input_values_vec                       = {610, 240, 550, 420, 450, 210, 220, 380, 370, 510, 530, 451, 840};

    const MoreSignificantKeyT max_value_of_more_significant_key = *std::max_element(std::begin(more_significant_keys_vec),
                                                                                    std::end(more_significant_keys_vec));
    const LessSignificantKeyT max_value_of_less_significant_key = *std::max_element(std::begin(less_significant_keys_vec),
                                                                                    std::end(less_significant_keys_vec));

    DefaultDeviceAllocator allocator = create_default_device_allocator();
    device_buffer<MoreSignificantKeyT> more_significant_keys(more_significant_keys_vec.size(), allocator);
    cudautils::device_copy_n(more_significant_keys_vec.data(), more_significant_keys_vec.size(), more_significant_keys.data()); //H2D
    device_buffer<LessSignificantKeyT> less_significant_keys(less_significant_keys_vec.size(), allocator);
    cudautils::device_copy_n(less_significant_keys_vec.data(), less_significant_keys_vec.size(), less_significant_keys.data()); //H2D
    device_buffer<ValueT> input_values(input_values_vec.size(), allocator);
    cudautils::device_copy_n(input_values_vec.data(), input_values_vec.size(), input_values.data()); //H2D

    test_function(more_significant_keys,
                  less_significant_keys,
                  input_values,
                  max_value_of_more_significant_key,
                  max_value_of_less_significant_key);
}

// repreat this test with differnt combinations of types, less significant key has larger max value than more significant key
template <typename MoreSignificantKeyT,
          typename LessSignificantKeyT,
          typename ValueT>
void short_test_template_larger_less_significant_key()
{
    // more less value
    //    6   10   610
    //    2   40   240
    //    5   50   550
    //    4   20   420
    //    4   50   450
    //    2   10   210
    //    2   20   220
    //    3   80   380
    //    3   70   370
    //    5   10   510
    //    5   30   530
    //    4   50   451
    //    8   40   840
    const std::vector<MoreSignificantKeyT> more_significant_keys_vec = {6, 2, 5, 4, 4, 2, 2, 3, 3, 5, 5, 4, 8};
    const std::vector<LessSignificantKeyT> less_significant_keys_vec = {10, 40, 50, 20, 50, 10, 20, 80, 70, 10, 30, 50, 40};
    const std::vector<ValueT> input_values_vec                       = {610, 240, 550, 420, 450, 210, 220, 380, 370, 510, 530, 451, 840};

    const MoreSignificantKeyT max_value_of_more_significant_key = *std::max_element(std::begin(more_significant_keys_vec),
                                                                                    std::end(more_significant_keys_vec));
    const LessSignificantKeyT max_value_of_less_significant_key = *std::max_element(std::begin(less_significant_keys_vec),
                                                                                    std::end(less_significant_keys_vec));

    DefaultDeviceAllocator allocator = create_default_device_allocator();
    device_buffer<MoreSignificantKeyT> more_significant_keys(more_significant_keys_vec.size(), allocator);
    cudautils::device_copy_n(more_significant_keys_vec.data(), more_significant_keys_vec.size(), more_significant_keys.data()); //H2D
    device_buffer<LessSignificantKeyT> less_significant_keys(less_significant_keys_vec.size(), allocator);
    cudautils::device_copy_n(less_significant_keys_vec.data(), less_significant_keys_vec.size(), less_significant_keys.data()); //H2D
    device_buffer<ValueT> input_values(input_values_vec.size(), allocator);
    cudautils::device_copy_n(input_values_vec.data(), input_values_vec.size(), input_values.data()); //H2D

    test_function(more_significant_keys,
                  less_significant_keys,
                  input_values,
                  max_value_of_more_significant_key,
                  max_value_of_less_significant_key);
}

TEST(TestUtilsCudasort, short_32_32_32_test)
{
    short_test_template_larger_more_significant_key<std::uint32_t, std::uint32_t, std::uint32_t>();
    short_test_template_larger_less_significant_key<std::uint32_t, std::uint32_t, std::uint32_t>();
}

TEST(TestUtilsCudasort, short_32_32_64_test)
{
    short_test_template_larger_more_significant_key<std::uint32_t, std::uint32_t, std::uint64_t>();
    short_test_template_larger_less_significant_key<std::uint32_t, std::uint32_t, std::uint64_t>();
}

TEST(TestUtilsCudasort, short_32_64_32_test)
{
    short_test_template_larger_more_significant_key<std::uint32_t, std::uint64_t, std::uint32_t>();
    short_test_template_larger_less_significant_key<std::uint32_t, std::uint64_t, std::uint32_t>();
}

TEST(TestUtilsCudasort, short_32_64_64_test)
{
    short_test_template_larger_more_significant_key<std::uint32_t, std::uint64_t, std::uint64_t>();
    short_test_template_larger_less_significant_key<std::uint32_t, std::uint64_t, std::uint64_t>();
}

TEST(TestUtilsCudasort, short_64_32_32_test)
{
    short_test_template_larger_more_significant_key<std::uint64_t, std::uint32_t, std::uint32_t>();
    short_test_template_larger_less_significant_key<std::uint64_t, std::uint32_t, std::uint32_t>();
}

TEST(TestUtilsCudasort, short_64_32_64_test)
{
    short_test_template_larger_more_significant_key<std::uint64_t, std::uint32_t, std::uint64_t>();
    short_test_template_larger_less_significant_key<std::uint64_t, std::uint32_t, std::uint64_t>();
}

TEST(TestUtilsCudasort, short_64_64_32_test)
{
    short_test_template_larger_more_significant_key<std::uint64_t, std::uint64_t, std::uint32_t>();
    short_test_template_larger_less_significant_key<std::uint64_t, std::uint64_t, std::uint32_t>();
}

TEST(TestUtilsCudasort, short_64_64_64_test)
{
    short_test_template_larger_more_significant_key<std::uint64_t, std::uint64_t, std::uint64_t>();
    short_test_template_larger_less_significant_key<std::uint64_t, std::uint64_t, std::uint64_t>();
}

TEST(TestUtilsCudasort, long_deterministic_shuffle_test)
{
    std::int64_t number_of_elements = 10'000'000;

    std::mt19937 g(10);

    DefaultDeviceAllocator allocator = create_default_device_allocator();

    // fill the arrays with values 0..number_of_elements and shuffle them
    thrust::host_vector<std::uint32_t> more_significant_keys_h(number_of_elements);
    std::iota(std::begin(more_significant_keys_h), std::end(more_significant_keys_h), 0);
    std::shuffle(std::begin(more_significant_keys_h), std::end(more_significant_keys_h), g);
    device_buffer<std::uint32_t> more_significant_keys_d(more_significant_keys_h.size(), allocator);
    cudautils::device_copy_n(more_significant_keys_h.data(), more_significant_keys_h.size(), more_significant_keys_d.data()); //H2D

    thrust::host_vector<std::uint32_t> less_significant_keys_h(number_of_elements);
    std::iota(std::begin(less_significant_keys_h), std::end(less_significant_keys_h), 0);
    std::shuffle(std::begin(less_significant_keys_h), std::end(less_significant_keys_h), g);
    device_buffer<std::uint32_t> less_significant_keys_d(less_significant_keys_h.size(), allocator);
    cudautils::device_copy_n(less_significant_keys_h.data(), less_significant_keys_h.size(), less_significant_keys_d.data()); //H2D

    thrust::host_vector<std::uint64_t> input_values_h(number_of_elements);
    std::transform(std::begin(more_significant_keys_h),
                   std::end(more_significant_keys_h),
                   std::begin(less_significant_keys_h),
                   std::begin(input_values_h),
                   [number_of_elements](const std::uint32_t more_significant_key, const std::uint32_t less_significant_key) {
                       return number_of_elements * more_significant_key + less_significant_key;
                   });

    device_buffer<std::uint64_t> input_values_d(input_values_h.size(), allocator);
    cudautils::device_copy_n(input_values_h.data(), input_values_h.size(), input_values_d.data()); //H2D

    const std::uint32_t max_value_of_more_significant_key = number_of_elements - 1;
    const std::uint32_t max_value_of_less_significant_key = number_of_elements - 1;

    test_function(more_significant_keys_d,
                  less_significant_keys_d,
                  input_values_d,
                  max_value_of_more_significant_key,
                  max_value_of_less_significant_key);
}

} // namespace genomeworks

} // namespace claraparabricks
